#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <omp.h>

#include <algorithm>
#include <cstddef>
#include <iostream>
#include <random>
#include <utility>
#include <vector>

using Matrix = std::vector<float>;
using Batch = std::vector<Matrix>;
using d_Batch = std::vector<float *>;
constexpr auto BLOCK_SIZE = 1024;

Matrix generateRandomMatrix(const size_t n) {
  static std::random_device rd;
  static std::mt19937 rng(rd());
  std::uniform_real_distribution<float> dis(0.0f, 1.0f);

  Matrix matrix;
  matrix.reserve(n * n);

  for (size_t i = 0; i < n * n; ++i) {
    matrix.emplace_back(dis(rng));
  }

  return matrix;
}

Batch generateRandomBatch(const size_t m, const size_t n) {
  Batch batch;
  batch.reserve(m);

  for (size_t i = 0; i < m; ++i) {
    batch.emplace_back(generateRandomMatrix(n));
  }

  return batch;
}

Batch convLayerSeq(const Batch &input_batch, const Matrix &filter,
                   const size_t m, const size_t n, const size_t k) {
  const auto c = n - k + 1;

  Batch output_batch;
  output_batch.reserve(m);

  for (const auto &input_matrix : input_batch) {
    Matrix output_matrix;
    output_matrix.reserve(c * c);

    for (size_t i = 0; i < c; ++i) {
      for (size_t j = 0; j < c; ++j) {
        float val = 0.0f;
        for (size_t e = 0; e < k; ++e) {
          for (size_t f = 0; f < k; ++f) {
            val += filter[e * k + f] * input_matrix[(i + e) * n + j + f];
          }
        }
        output_matrix.emplace_back(val);
      }
    }

    output_batch.emplace_back(std::move(output_matrix));
  }

  return output_batch;
}

Batch convLayerPar(const Batch &input_batch, const Matrix &filter,
                   const size_t m, const size_t n, const size_t k) {
  const auto c = n - k + 1;
  Batch output_batch(m);

#pragma omp parallel for
  for (size_t b = 0; b < m; ++b) {
    const auto &input_matrix = input_batch[b];
    Matrix output_matrix;
    output_matrix.reserve(c * c);

    for (size_t i = 0; i < c; ++i) {
      for (size_t j = 0; j < c; ++j) {
        float val = 0.0f;
        for (size_t e = 0; e < k; ++e) {
          for (size_t f = 0; f < k; ++f) {
            val += filter[e * k + f] * input_matrix[(i + e) * n + j + f];
          }
        }
        output_matrix.emplace_back(val);
      }
    }

    output_batch[b] = std::move(output_matrix);
  }

  return output_batch;
}

float *matrixToDevice(const Matrix &matrix) {
  const auto matrix_size = matrix.size() * sizeof(float);
  float *d_matrix = nullptr;

  hipMalloc((void **)&d_matrix, matrix_size);
  hipMemcpy((void *)d_matrix, (void *)matrix.data(), matrix_size,
             hipMemcpyHostToDevice);

  return d_matrix;
}

d_Batch batchToDevice(const Batch &batch) {
  d_Batch d_batch;
  d_batch.reserve(batch.size());

  for (const auto &matrix : batch) {
    d_batch.emplace_back(matrixToDevice(matrix));
  }

  return d_batch;
}

d_Batch initDeviceBatch(const size_t bathc_size, const size_t n) {
  const auto matrix_size = n * n * sizeof(float);
  d_Batch d_batch(bathc_size);

  for (const auto &d_matrix : d_batch) {
    hipMalloc((void **)d_matrix, matrix_size);
  }

  return d_batch;
}

void freeDeviceBatch(d_Batch &d_batch) {
  for (auto &d_matrix : d_batch) {
    hipFree(d_matrix);
  }
}

Batch batchToHost(const d_Batch &d_batch, const size_t n) {
  Batch batch;
  batch.reserve(d_batch.size());

  for (const auto &d_matrix : d_batch) {
    Matrix matrix(n * n);
    hipMemcpy(matrix.data(), d_matrix, n * n * sizeof(float),
               hipMemcpyDeviceToHost);
    batch.emplace_back(std::move(matrix));
  }

  return batch;
}

__global__ void convLayerKernel(const float *input_matrix,
                                float *output_matrix, const float *filter,
                                const size_t n, const size_t c,
                                const size_t k) {
  __shared__ float filter_cache[BLOCK_SIZE];

  const auto row = blockIdx.x;
  const auto col = threadIdx.x;

  float val = 0.0f;
  for (size_t e = 0; e < k; ++e) {
    for (size_t f = 0; f < k; ++f) {
      val += filter[e * k + f] * input_matrix[(row + e) * n + col + f];
    }
  }
  output_matrix[row * c + col] = val;

}

std::pair<Batch, float> convLayerCUDA(const Batch &input_batch,
                                      const Matrix &filter, const size_t m,
                                      const size_t n, const size_t k) {
  const auto c = n - k + 1;
  auto d_input_batch = batchToDevice(input_batch);
  auto d_filter = matrixToDevice(filter);
  auto d_output_batch = initDeviceBatch(m, c);

  const auto blocks_per_grid = n;
  const auto threads_per_block = n;
  const auto shared_memory_size = k * k * sizeof(float);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  for (size_t i = 0; i < m; ++i) {
    convLayerKernel<<<blocks_per_grid, threads_per_block, shared_memory_size>>>(
        d_input_batch[i], d_output_batch[i], d_filter, n, c, k);
  }

  hipEventSynchronize(stop);
  hipEventRecord(stop);

  float sec = 0.0f;
  hipEventElapsedTime(&ms, start, stop);
  sec /= 1000.0f;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  const auto output_batch = batchToDevice(d_output_batch);

  freeDeviceBatch(d_input_batch);
  freeDeviceBatch(d_output_batch);
  hipFree(d_filter);

  return {output_batch, sec};
}

float maxDifference(const Batch &batch1, const Batch &batch2) {
  float max_difference = 0.0f;

  for (size_t i = 0; i < batch1.size(); ++i) {
    const auto &m1 = batch1[i];
    const auto &m2 = batch2[i];

    for (size_t j = 0; j < m1.size(); ++j) {
      const auto difference = std::abs(m1[j] - m2[j]);
      if (difference > max_difference) {
        max_difference = difference;
      }
    }
  }

  return max_difference;
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    std::cerr << "Usage: " << argv[0]
              << " <matrix_size> <filter_size> <batch_size>" << std::endl;
    return 1;
  }
  const auto n = std::stoull(argv[1]);
  const auto k = std::stoull(argv[2]);
  const auto m = std::stoull(argv[3]);

  try {
    const auto batch = generateRandomBatch(m, n);
    const auto filter = generateRandomMatrix(k);

    const auto seq_start = omp_get_wtime();
    const auto seq_result = convLayerSeq(batch, filter, m, n, k);
    const auto seq_end = omp_get_wtime();
    std::cout << "Sequential time: " << seq_end - seq_start << std::endl;

    {
      const auto par_start = omp_get_wtime();
      const auto par_result = convLayerPar(batch, filter, m, n, k);
      const auto par_end = omp_get_wtime();
      std::cout << "Parallel time: " << par_end - par_start << " ";
      std::cout << "(diff: " << maxDifference(seq_result, par_result) << ")"
                << std::endl;
    }

    {
      const auto [cuda_result, cuda_time] =
          convLayerCUDA(batch, filter, m, n, k);
      std::cout << "Cuda time: " << cuda_time << " ";
      std::cout << "(diff: " << maxDifference(seq_result, cuda_result) << ")"
                << std::endl;
    }

  } catch (const std::bad_alloc &e) {
    std::cerr << "Error alloc memory: " << e.what() << std::endl;
    return 1;
  }

  return 0;
}
